#include "hip/hip_runtime.h"
#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include <iostream>

#include "body.h"
#include "motion.h"
#include "utils.h"

// gravitational constant
const float G = 6.674E-11;
const float epsilon = .25f * .25f;

const int BLOCK_THREADS = 1024;//512;
#if __CUDA_ARCH__ >= 200
    const int maxThreadsPerBlock = 1024;
    const int minBlocks = 2;
#else
    const int maxThreadsPerBlock = 512 / 2;
    const int minBlocks = 2;
#endif

void run_calculations(int N, float timestep, 
                        hipStream_t stream1, hipStream_t stream2, float* mass,
                        float* posx, float* posy, float* posz, float* velx, 
                        float* vely, float* velz, float* accx, float* accy,
                        float* accz)
{
    int blocks = N / BLOCK_THREADS;
    blocks = N % BLOCK_THREADS == 0 ? blocks : blocks + 1;

    calculate_force<<<blocks, BLOCK_THREADS, sizeof(float) * 4, stream1>>>(N,  timestep,
                mass, posx, posy, posz, accx, accy, accz);
    #ifdef cudaerr
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    #endif
    update_pos<<<blocks, BLOCK_THREADS, 0, stream2>>>(N, posx, posy,
                posz, velx, vely, velz, accx, accy, accz);
    #ifdef cudaerr
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    #endif
}

extern "C" __global__
//__launch_bounds__(maxThreadsPerBlock, minBlocks)
void calculate_force(int nbodies, float timestep, float* gmass,
                     float* posx, float* posy, float* posz, float* accx, float*
                     accy, float* accz)
{
    float dx;
    float dy;
    float dz;
    float dist;
    float mass;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    #pragma unroll 16
    for (int j = 0; j < nbodies; j++)
    {
        //__shared__ float x, y, z, mass;
        //x = posx[j];
        //y = posy[j];
        //z = posz[j];
    //    mass = timestep * G;
//        __syncthreads();
        if (j != i)
        {
            //dx = x - posx[i];
            //dy = y - posy[i];
            //dz = z - posz[i];
            dx = posx[j] - posx[i];
            dy = posy[j] - posy[i];
            dz = posz[j] - posz[i];

            //dist = sqrt(dx * dx + dy * dy + dz * dz);
            //dist = dist * dist * dist;
            dist = (dx * dx) + ((dy * dy) + (dz * dz));
            dist = (1.0f/dist) * sqrtf(dist) + epsilon;
            //dist = dist < 1.0f ? dist : 1.0f;
            //omit mass of current body, must divide by it later to get accel anyway
            mass = timestep * G * gmass[j];
      //      mass = mass * gmass[j];

            accx[i] += mass * dx * dist;
            accy[i] += mass * dy * dist;
            accz[i] += mass * dz * dist;
        }
    }
}

extern "C" __global__
//__launch_bounds__(maxThreadsPerBlock, minBlocks)
void update_pos(int nbodies, 
                     float* posx, float* posy, float* posz, float* velx, float*
                     vely, float* velz, float* accx, float* accy, float* accz)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    velx[i] += accx[i];
    vely[i] += accy[i];
    velz[i] += accz[i];
    posx[i] += velx[i];
    posy[i] += vely[i];
    posz[i] += velz[i];
}

#include "hip/hip_runtime.h"
#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include <iostream>

#include "body.h"
#include "motion.h"
#include "utils.h"

// gravitational constant
const float G = 6.674E-11;
const float epsilon = .25f * .25f;

const int BLOCK_THREADS = 256;//1024;

void run_calculations(int N, Body* g_bodies, float timestep, hipStream_t stream1, hipStream_t stream2)
{
    int blocks = N / BLOCK_THREADS;
    blocks = N % BLOCK_THREADS == 0 ? blocks : blocks + 1;

    calculate_force<<<blocks, BLOCK_THREADS, 0, stream1>>>(N, g_bodies, timestep);
    #ifdef cudaerr
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    #endif
    update_pos<<<blocks, BLOCK_THREADS, 0, stream2>>>(N, g_bodies);
    #ifdef cudaerr
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    #endif
}

extern "C" __global__
void calculate_force(int nbodies, Body* g_bodies, float timestep)
{
    float dx;
    float dy;
    float dz;
    float dist;
    float mass;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    for (int j = 0; j < nbodies; j++)
    {
        if (j != i)
        {
            dx = g_bodies[j].get_pos(0) - g_bodies[i].get_pos(0);
            dy = g_bodies[j].get_pos(1) - g_bodies[i].get_pos(1);
            dz = g_bodies[j].get_pos(2) - g_bodies[i].get_pos(2);


            //dist = sqrt(dx * dx + dy * dy + dz * dz);
            //dist = dist * dist * dist;
            dist = dx * dx + dy * dy + dz * dz;
            dist = (1.0f/dist) * sqrtf(dist) + epsilon;
            //dist = dist < 1.0f ? dist : 1.0f;
            //omit mass of current body, must divide by it later to get accel anyway
            mass = timestep * G * g_bodies[j].get_mass();

            //g_bodies[i].update_acc(0, mass * dx / dist);
            //g_bodies[i].update_acc(1, mass * dy / dist);
            //g_bodies[i].update_acc(2, mass * dz / dist);
            g_bodies[i].update_acc(0, mass * dx * dist);
            g_bodies[i].update_acc(1, mass * dy * dist);
            g_bodies[i].update_acc(2, mass * dz * dist);
        }
    }
}

extern "C" __global__
void update_pos(int nbodies, Body* g_bodies)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    g_bodies[i].update_vel(0, g_bodies[i].get_acc(0));
    g_bodies[i].update_vel(1, g_bodies[i].get_acc(1));
    g_bodies[i].update_vel(2, g_bodies[i].get_acc(2));
    g_bodies[i].update_pos(0, g_bodies[i].get_vel(0));
    g_bodies[i].update_pos(1, g_bodies[i].get_vel(1));
    g_bodies[i].update_pos(2, g_bodies[i].get_vel(2));
}

#include "body.h"

#include <sstream>
#include <string>
#include <iostream>

Body::Body() : Body::Body(.01, .75) {}

Body::Body(float mass, float radius)
{
    this->mass = mass;
    this->radius = radius;
    body = gluNewQuadric();

    gluQuadricDrawStyle(body, GLU_FILL);
    gluQuadricNormals(body, GLU_SMOOTH);
    gluQuadricOrientation(body, GLU_OUTSIDE);
    gluQuadricTexture(body, GL_TRUE);

    for (size_t i = 0; i < DIMS; i++)
    {
        set_pos(i, 0);                 
        set_vel(i, 0);                 
        set_acc(i, 0);                 
        set_color(i, 1.0);
    }
//    set_pos(3, 1);
//    set_color(3, 1);
}

extern "C" __device__ __host__
void Body::set_mass(float mass)
{
    this->mass = mass;
}

extern "C" __device__ __host__
void Body::set_radius(float radius)
{
    this->radius = radius;
}

extern "C" __device__ __host__
float Body::get_color(size_t idx)
{
    return color[idx];
}

extern "C" __device__ __host__
float Body::get_mass()
{
    return mass;
}

extern "C" __device__ __host__
float Body::get_radius()
{
    return radius;
}

extern "C" __device__ __host__
float Body::get_pos(size_t idx)
{
    return pos[idx];
}

extern "C" __device__ __host__
float Body::get_vel(size_t idx)
{
    return vel[idx];
}

extern "C" __device__ __host__
float Body::get_acc(size_t idx)
{
    return acc[idx];
}

extern "C" __device__ __host__
void Body::set_pos(size_t idx, float val)
{
    pos[idx] = val;
}

extern "C" __device__ __host__
void Body::set_vel(size_t idx, float val)
{
    vel[idx] = val;
}

extern "C" __device__ __host__
void Body::set_acc(size_t idx, float val)
{
    acc[idx] = val;
}

extern "C" __device__ __host__
void Body::set_color(size_t idx, float val)
{
    color[idx] = val;
}

extern "C" __device__ __host__
void Body::update_pos(size_t idx, float delta)
{
    pos[idx] += delta;
}

extern "C" __device__ __host__
void Body::update_vel(size_t idx, float delta)
{
    vel[idx] += delta;
}

extern "C" __device__ __host__
void Body::update_acc(size_t idx, float delta)
{
    acc[idx] += delta;
}

std::string Body::to_string()
{
    size_t i;
    std::ostringstream os;
    os << "(mass: " << mass << ", radius: " << radius << ", ";
    for (i = 0; i < DIMS; i++)
    {
        os << "color[" << i << "]: " << color[i] << ", ";
    }
    for (i = 0; i < DIMS; i++)
    {
        os << "pos[" << i << "]: " << pos[i] << ", ";
    }
    for (i = 0; i < DIMS; i++)
    {
        os << "vel[" << i << "]: " << vel[i] << ", ";
    }
    for (i = 0; i < DIMS - 1; i++)
    {
        os << "acc[" << i << "]: " << acc[i] << ", ";
    }
    os << "acc[" << i << "]: " << acc[i] << ")";
    return os.str();
}


#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <>

#include <hip/hip_runtime_api.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <helper_functions.h>
#include <helper_cuda_gl.h>

#include "body.h"
#include "motion.h"

void simulate(int N, Body* g_bodies);
extern "C" __global__ void calculate_force(int, Body*);
extern "C" __global__ void update_pos(int, Body*);

// gravitational constant
const double G = 6.674E-11;

//seconds, for now.
const double timestep = .0001;

const int BLOCK_THREADS = 2048;

void simulate(int N, Body* g_bodies)
{
    int blocks = N / BLOCK_THREADS;   
    calculate_force<<<blocks, BLOCK_THREADS>>>(N, g_bodies);
    update_pos<<<blocks, BLOCK_THREADS>>>(N, g_bodies);
}

extern "C" __global__
void calculate_force(int nbodies, Body* g_bodies)
{
    double dx;
    double dy;
    double dz;
    double dist;
    double mass;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < nbodies)
    {
        for (int j = 0; j < nbodies; j++)
        {
            if (i != j)
            {
                dx = g_bodies[j].get_pos(0) - g_bodies[i].get_pos(0);
                dy = g_bodies[j].get_pos(1) - g_bodies[i].get_pos(1);
                dz = g_bodies[j].get_pos(2) - g_bodies[i].get_pos(2);
                dist = sqrt(dx * dx + dy * dy + dz * dz);
                dist = dist * dist * dist;
                //omit mass of current body, must divide by it later to get accel anyway
                mass = timestep * G * g_bodies[j].get_mass();
                g_bodies[i].update_acc(0, mass * dx / dist);
                g_bodies[i].update_acc(1, mass * dx / dist);
                g_bodies[i].update_acc(2, mass * dx / dist);
            }
        }
    }
}

extern "C" __global__
void update_pos(int nbodies, Body* g_bodies)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    g_bodies[i].update_vel(0, g_bodies[i].get_acc(0));
    g_bodies[i].update_vel(1, g_bodies[i].get_acc(1));
    g_bodies[i].update_vel(2, g_bodies[i].get_acc(2));
    g_bodies[i].update_acc(0, g_bodies[i].get_vel(0));
    g_bodies[i].update_acc(1, g_bodies[i].get_vel(1));
    g_bodies[i].update_acc(2, g_bodies[i].get_vel(2));
}

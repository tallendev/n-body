#include "hip/hip_runtime.h"
#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include <iostream>

#include "body.h"
#include "motion.h"
#include "utils.h"

// gravitational constant
const double G = 1;//6.674E-11;

const int BLOCK_THREADS = 1024; //2048;

void run_calculations(int N, Body* g_bodies, double timestep)
{
    int blocks = N / BLOCK_THREADS;
    blocks = blocks == 0 ? 1 : blocks;   
    calculate_force<<<blocks, BLOCK_THREADS>>>(N, g_bodies, timestep);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    update_pos<<<blocks, BLOCK_THREADS>>>(N, g_bodies);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

extern "C" __global__
void calculate_force(int nbodies, Body* g_bodies, double timestep)
{
    double dx;
    double dy;
    double dz;
    double dist;
    double mass;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < nbodies)
    {
        for (int j = 0; j < nbodies; j++)
        {
            if (i != j)
            {
                dx = g_bodies[j].get_pos(0) - g_bodies[i].get_pos(0);
                dy = g_bodies[j].get_pos(1) - g_bodies[i].get_pos(1);
                dz = g_bodies[j].get_pos(2) - g_bodies[i].get_pos(2);
                dist = sqrt(dx * dx + dy * dy + dz * dz);
                dist = dist * dist * dist;
                //omit mass of current body, must divide by it later to get accel anyway
                mass = timestep * G * g_bodies[j].get_mass();
                g_bodies[i].update_acc(0, mass * dx / dist);
                g_bodies[i].update_acc(1, mass * dy / dist);
                g_bodies[i].update_acc(2, mass * dz / dist);
            }
        }
    }
}

extern "C" __global__
void update_pos(int nbodies, Body* g_bodies)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    g_bodies[i].update_vel(0, g_bodies[i].get_acc(0));
    g_bodies[i].update_vel(1, g_bodies[i].get_acc(1));
    g_bodies[i].update_vel(2, g_bodies[i].get_acc(2));
    g_bodies[i].update_pos(0, g_bodies[i].get_vel(0));
    g_bodies[i].update_pos(1, g_bodies[i].get_vel(1));
    g_bodies[i].update_pos(2, g_bodies[i].get_vel(2));
}

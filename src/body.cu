#include "body.h"

#include <sstream>
#include <string>
#include <iostream>

Body::Body() : Body::Body(.01, .75) {}

Body::Body(double mass, double radius)
{
    this->mass = mass;
    this->radius = radius;
    body = gluNewQuadric();

    gluQuadricDrawStyle(body, GLU_FILL);
    gluQuadricNormals(body, GLU_SMOOTH);
    gluQuadricOrientation(body, GLU_OUTSIDE);
    gluQuadricTexture(body, GL_TRUE);

    for (size_t i = 0; i < DIMS; i++)
    {
        set_pos(i, 0);                 
        set_vel(i, 0);                 
        set_acc(i, 0);                 
        set_color(i, 1.0);
    }
//    set_pos(3, 1);
//    set_color(3, 1);
}

extern "C" __device__ __host__
void Body::set_mass(double mass)
{
    this->mass = mass;
}

extern "C" __device__ __host__
void Body::set_radius(double radius)
{
    this->radius = radius;
}

extern "C" __device__ __host__
double Body::get_color(size_t idx)
{
    return color[idx];
}

extern "C" __device__ __host__
double Body::get_mass()
{
    return mass;
}

extern "C" __device__ __host__
double Body::get_radius()
{
    return radius;
}

extern "C" __device__ __host__
double Body::get_pos(size_t idx)
{
    return pos[idx];
}

extern "C" __device__ __host__
double Body::get_vel(size_t idx)
{
    return vel[idx];
}

extern "C" __device__ __host__
double Body::get_acc(size_t idx)
{
    return acc[idx];
}

extern "C" __device__ __host__
void Body::set_pos(size_t idx, double val)
{
    pos[idx] = val;
}

extern "C" __device__ __host__
void Body::set_vel(size_t idx, double val)
{
    vel[idx] = val;
}

extern "C" __device__ __host__
void Body::set_acc(size_t idx, double val)
{
    acc[idx] = val;
}

extern "C" __device__ __host__
void Body::set_color(size_t idx, double val)
{
    color[idx] = val;
}

extern "C" __device__ __host__
void Body::update_pos(size_t idx, double delta)
{
    pos[idx] += delta;
}

extern "C" __device__ __host__
void Body::update_vel(size_t idx, double delta)
{
    vel[idx] += delta;
}

extern "C" __device__ __host__
void Body::update_acc(size_t idx, double delta)
{
    acc[idx] += delta;
}

/*
void Body::render()
{
    glColor3f(color[0], color[1], color[2]);
    glTranslated(pos[X], pos[Y], pos[Z]);
    gluSphere(body, radius, slices, stacks); 
}
*/

std::string Body::to_string()
{
    size_t i;
    std::ostringstream os;
    os << "(mass: " << mass << ", radius: " << radius << ", ";
    for (i = 0; i < DIMS; i++)
    {
        os << "color[" << i << "]: " << color[i] << ", ";
    }
    for (i = 0; i < DIMS; i++)
    {
        os << "pos[" << i << "]: " << pos[i] << ", ";
    }
    for (i = 0; i < DIMS; i++)
    {
        os << "vel[" << i << "]: " << vel[i] << ", ";
    }
    for (i = 0; i < DIMS - 1; i++)
    {
        os << "acc[" << i << "]: " << acc[i] << ", ";
    }
    os << "acc[" << i << "]: " << acc[i] << ")";
    return os.str();
}

